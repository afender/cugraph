/*
 * Copyright (c) 2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

// Graph tests
// Author: Alex Fender afender@nvidia.com

#include "gtest/gtest.h"
#include "types.h"
#include "functions.h"
#include "test_utils.h"
#include <string.h>
#include <rmm_utils.h>

TEST(edge_list, size_mismatch)
{
       
  typedef int VT;
  typedef float WT;

  Graph_ptr<VT,WT> G{new cugraph::Graph<VT,WT>, Graph_deleter<VT,WT>};
  
  std::vector<VT> src_h={0, 0, 2, 2, 2, 3, 3, 4, 4, 5}, dest_h={1, 2, 0, 1, 4};
  std::vector<WT> w_h={0.50, 0.50, 0.33, 0.33, 0.33, 0.50, 0.50, 0.50, 0.50};

  d_ptr<VT> d_src = create_d_ptr(src_h);
  d_ptr<VT> d_dst = create_d_ptr(dest_h);
  d_ptr<WT> d_w = create_d_ptr(w_h);

  ASSERT_THROW(cugraph::edge_list_view(G.get(), d_src.get(), d_dst.get(), d_w.get()), std::logic_error);
}


TEST(edge_list, size_mismatch2)
{
  typedef int VT;
  typedef float WT;
       
  Graph_ptr<VT,WT> G{new cugraph::Graph<VT,WT>, Graph_deleter<VT,WT>};
  
  std::vector<VT> src_h={0, 0, 2, 2, 2, 3, 3, 4, 4, 5}, dest_h={1, 2, 0, 1, 4, 4, 5, 3, 5, 3};
  std::vector<WT> w_h={0.50, 0.50, 0.33, 0.33, 0.33, 0.50, 0.50, 0.50};
  
  d_ptr<VT> d_src = create_d_ptr(src_h);
  d_ptr<VT> d_dst = create_d_ptr(dest_h);
  d_ptr<WT> d_w = create_d_ptr(w_h);

  ASSERT_THROW(cugraph::edge_list_view(G.get(), d_src.get(), d_dst.get(), d_w.get()), std::logic_error);

}

TEST(adj_list, success)
{
  typedef int VT;
  typedef float WT;
  // Hard-coded Zachary Karate Club network input
  std::vector<VT> off_h = {0, 16, 25, 35, 41, 44, 48, 52, 56, 61, 63, 66, 67, 69, 74, 76, 78, 80, 82, 84, 87, 89, 91, 93, 98, 101, 104, 106, 110, 113, 117, 121, 127, 
      139, 156};
  std::vector<VT> ind_h = {1, 2, 3, 4, 5, 6, 7, 8, 10, 11, 12, 13, 17, 19, 21, 31, 0, 2, 3, 7, 13, 17, 19, 21, 30, 0, 1, 3, 7, 8, 9, 13, 27, 28, 32, 0, 1, 2, 7, 12, 13, 0, 6, 10, 0, 
      6, 10, 16, 0, 4, 5, 16, 0, 1, 2, 3, 0, 2, 30, 32, 33, 2, 33, 0, 4, 5, 0, 0, 3, 0, 1, 2, 3, 33, 32, 33, 32, 33, 5, 6, 0, 1, 32, 33, 0, 1, 33, 32, 33, 0, 1, 32, 33, 25, 27, 29, 32, 33, 
      25, 27, 31, 23, 24, 31, 29, 33, 2, 23, 24, 33, 2, 31, 33, 23, 26, 32, 33, 1, 8, 32, 33, 0, 24, 25, 28, 32, 33, 2, 8, 14, 15, 18, 20, 22, 23, 29, 30, 31, 33, 8, 9, 13, 14, 15, 
      18, 19, 20, 22, 23, 26, 27, 28, 29, 30, 31, 32};
  std::vector<WT> w_h = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
      
  Graph_ptr<VT,WT> G{new cugraph::Graph<VT,WT>, Graph_deleter<VT,WT>};
  
  d_ptr<VT> d_off = create_d_ptr(off_h);
  d_ptr<VT> d_ind = create_d_ptr(ind_h);
  d_ptr<WT> d_w = create_d_ptr(w_h);

  cugraph::adj_list_view(G.get(), d_off.get(), d_ind.get(), d_w.get());

  std::vector<VT> off2_h(off_h.size()), ind2_h(ind_h.size());
  std::vector<WT> w2_h(w_h.size());

  hipMemcpy(&off2_h[0], G.get()->adjList->offsets, sizeof(VT) * off_h.size(), hipMemcpyDeviceToHost);
  hipMemcpy(&ind2_h[0], G.get()->adjList->indices, sizeof(VT) * ind_h.size(), hipMemcpyDeviceToHost);
  hipMemcpy(&w2_h[0], G.get()->adjList->edge_data, sizeof(WT) * w_h.size(), hipMemcpyDeviceToHost);
  
  ASSERT_EQ( eq(off_h,off2_h), 0);
  ASSERT_EQ( eq(ind_h,ind2_h), 0);
  ASSERT_EQ( eq(w_h,w2_h), 0);
}

TEST(adj_list, success_no_weights)
{
  typedef int VT;
  // Hard-coded Zachary Karate Club network input
  std::vector<VT> off_h = {0, 16, 25, 35, 41, 44, 48, 52, 56, 61, 63, 66, 67, 69, 74, 76, 78, 80, 82, 84, 87, 89, 91, 93, 98, 101, 104, 106, 110, 113, 117, 121, 127, 
      139, 156};
  std::vector<VT> ind_h = {1, 2, 3, 4, 5, 6, 7, 8, 10, 11, 12, 13, 17, 19, 21, 31, 0, 2, 3, 7, 13, 17, 19, 21, 30, 0, 1, 3, 7, 8, 9, 13, 27, 28, 32, 0, 1, 2, 7, 12, 13, 0, 6, 10, 0, 
      6, 10, 16, 0, 4, 5, 16, 0, 1, 2, 3, 0, 2, 30, 32, 33, 2, 33, 0, 4, 5, 0, 0, 3, 0, 1, 2, 3, 33, 32, 33, 32, 33, 5, 6, 0, 1, 32, 33, 0, 1, 33, 32, 33, 0, 1, 32, 33, 25, 27, 29, 32, 33, 
      25, 27, 31, 23, 24, 31, 29, 33, 2, 23, 24, 33, 2, 31, 33, 23, 26, 32, 33, 1, 8, 32, 33, 0, 24, 25, 28, 32, 33, 2, 8, 14, 15, 18, 20, 22, 23, 29, 30, 31, 33, 8, 9, 13, 14, 15, 
      18, 19, 20, 22, 23, 26, 27, 28, 29, 30, 31, 32};
      
  Graph_ptr<VT> G{new cugraph::Graph<VT>, Graph_deleter<VT>};
  
  d_ptr<VT> d_off = create_d_ptr(off_h);
  d_ptr<VT> d_ind = create_d_ptr(ind_h);

  cugraph::adj_list_view(G.get(), d_off.get(), d_ind.get());

  std::vector<VT> off2_h(off_h.size()), ind2_h(ind_h.size());

  hipMemcpy(&off2_h[0], G.get()->adjList->offsets, sizeof(VT) * off_h.size(), hipMemcpyDeviceToHost);
  hipMemcpy(&ind2_h[0], G.get()->adjList->indices, sizeof(VT) * ind_h.size(), hipMemcpyDeviceToHost);
  
  ASSERT_EQ( eq(off_h,off2_h), 0);
  ASSERT_EQ( eq(ind_h,ind2_h), 0);
}

TEST(Graph_properties, success)
{
  typedef int VT;
  typedef float WT;

  Graph_ptr<VT,WT> G{new cugraph::Graph<VT,WT>, Graph_deleter<VT,WT>};
  cugraph::Graph_properties *prop = new cugraph::Graph_properties;
  ASSERT_FALSE(prop->directed);
  ASSERT_FALSE(prop->weighted);
  ASSERT_FALSE(prop->multigraph);
  ASSERT_FALSE(prop->bipartite);
  ASSERT_FALSE(prop->tree);
  prop->directed = true;
  prop->weighted = true;
  prop->tree = false;
  ASSERT_TRUE(prop->directed);
  ASSERT_TRUE(prop->weighted);
  ASSERT_FALSE(prop->multigraph);
  ASSERT_FALSE(prop->bipartite);
  ASSERT_FALSE(prop->tree);
}

TEST(number_of_vertices, success1)
{
  typedef int VT;
  typedef float WT;

  std::vector<VT> src_h={0, 0, 2, 2, 2, 3, 3, 4, 4, 5};
  std::vector<VT> dest_h={1, 2, 0, 1, 4, 4, 5, 3, 5, 3};
  std::vector<WT> w_h={0.50, 0.50, 0.33, 0.33, 0.33, 0.50, 0.50, 0.50, 0.50, 0.5};

  cugraph::Graph G;
  gdf_column d_src, d_dst, d_w;
  create_d_ptr(src_h, &d_src);
  create_d_ptr(dest_h, &d_dst);
  create_d_ptr(w_h, &d_w);

  cugraph::edge_list_view(&G, &d_src, &d_dst, &d_w);
  ASSERT_EQ(G.numberOfVertices, 0);

  cugraph::number_of_vertices(&G);

  ASSERT_EQ(G.numberOfVertices, 6);
}

TEST(gdf_delete_adjacency_list, success1)
{
  typedef int VT;
  typedef float WT;
  // Hard-coded Zachary Karate Club network input
  std::vector<VT> off_h = {0, 16, 25, 35, 41, 44, 48, 52, 56, 61, 63, 66, 67, 69, 74, 76, 78, 80, 82, 84, 87, 89, 91, 93, 98, 101, 104, 106, 110, 113, 117, 121, 127, 
      139, 156};
  std::vector<VT> ind_h = {1, 2, 3, 4, 5, 6, 7, 8, 10, 11, 12, 13, 17, 19, 21, 31, 0, 2, 3, 7, 13, 17, 19, 21, 30, 0, 1, 3, 7, 8, 9, 13, 27, 28, 32, 0, 1, 2, 7, 12, 13, 0, 6, 10, 0, 
      6, 10, 16, 0, 4, 5, 16, 0, 1, 2, 3, 0, 2, 30, 32, 33, 2, 33, 0, 4, 5, 0, 0, 3, 0, 1, 2, 3, 33, 32, 33, 32, 33, 5, 6, 0, 1, 32, 33, 0, 1, 33, 32, 33, 0, 1, 32, 33, 25, 27, 29, 32, 33, 
      25, 27, 31, 23, 24, 31, 29, 33, 2, 23, 24, 33, 2, 31, 33, 23, 26, 32, 33, 1, 8, 32, 33, 0, 24, 25, 28, 32, 33, 2, 8, 14, 15, 18, 20, 22, 23, 29, 30, 31, 33, 8, 9, 13, 14, 15, 
      18, 19, 20, 22, 23, 26, 27, 28, 29, 30, 31, 32};
  std::vector<WT> w_h = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
      
  cugraph::Graph G;
  gdf_column d_off, d_ind, d_w;
  //size_t free, free2, total;  
  //hipMemGetInfo(&free, &total);
  create_d_ptr(off_h, &d_off);
  create_d_ptr(ind_h, &d_ind);
  create_d_ptr(w_h, &d_w);

  cugraph::adj_list_view(&G, &d_off, &d_ind, &d_w);
  
  //hipMemGetInfo(&free2, &total);
  //EXPECT_NE(free,free2);
  
  cugraph::delete_adj_list(&G);

  //hipMemGetInfo(&free2, &total);
  //EXPECT_EQ(free,free2);
}

TEST(gdf_delete_adjacency_list, success2)
{
  typedef int VT;
  typedef float WT;
  // Hard-coded Zachary Karate Club network input
  std::vector<VT> off_h = {0, 16, 25, 35, 41, 44, 48, 52, 56, 61, 63, 66, 67, 69, 74, 76, 78, 80, 82, 84, 87, 89, 91, 93, 98, 101, 104, 106, 110, 113, 117, 121, 127, 
      139, 156};
  std::vector<VT> ind_h = {1, 2, 3, 4, 5, 6, 7, 8, 10, 11, 12, 13, 17, 19, 21, 31, 0, 2, 3, 7, 13, 17, 19, 21, 30, 0, 1, 3, 7, 8, 9, 13, 27, 28, 32, 0, 1, 2, 7, 12, 13, 0, 6, 10, 0, 
      6, 10, 16, 0, 4, 5, 16, 0, 1, 2, 3, 0, 2, 30, 32, 33, 2, 33, 0, 4, 5, 0, 0, 3, 0, 1, 2, 3, 33, 32, 33, 32, 33, 5, 6, 0, 1, 32, 33, 0, 1, 33, 32, 33, 0, 1, 32, 33, 25, 27, 29, 32, 33, 
      25, 27, 31, 23, 24, 31, 29, 33, 2, 23, 24, 33, 2, 31, 33, 23, 26, 32, 33, 1, 8, 32, 33, 0, 24, 25, 28, 32, 33, 2, 8, 14, 15, 18, 20, 22, 23, 29, 30, 31, 33, 8, 9, 13, 14, 15, 
      18, 19, 20, 22, 23, 26, 27, 28, 29, 30, 31, 32};
  std::vector<WT> w_h = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
      
  cugraph::Graph *G = new cugraph::Graph;
  gdf_column *d_off = new gdf_column, *d_ind = new gdf_column, *d_w = new gdf_column;
  //size_t free, free2, total;  
  //hipMemGetInfo(&free, &total);
  create_d_ptr(off_h, d_off);
  create_d_ptr(ind_h, d_ind);
  create_d_ptr(w_h, d_w);

  cugraph::adj_list_view(G, d_off, d_ind, d_w);
  
  //hipMemGetInfo(&free2, &total);
  //EXPECT_NE(free,free2);
  
  cugraph::delete_adj_list(G);

  //hipMemGetInfo(&free2, &total);
  //EXPECT_EQ(free,free2);

  delete G;
  delete d_off;
  delete d_ind;
  delete d_w;
}


TEST(delete_edge_list, success1)
{
  typedef int VT;
  typedef float WT;

  std::vector<VT> src_h={0, 0, 2, 2, 2, 3, 3, 4, 4, 5}, dest_h={1, 2, 0, 1, 4, 4, 5, 3, 5, 3};
  std::vector<WT> w_h={0.50, 0.50, 0.33, 0.33, 0.33, 0.50, 0.50, 0.50, 0.50, 1.00};

  cugraph::Graph G ;
  gdf_column d_src, d_dst, d_w;
  //size_t free, free2, total;  
  //hipMemGetInfo(&free, &total);
  create_d_ptr(src_h, &d_src);
  create_d_ptr(dest_h, &d_dst);
  create_d_ptr(w_h, &d_w);

  cugraph::edge_list_view(&G, &d_src, &d_dst, &d_w);
  
  //hipMemGetInfo(&free2, &total);
  //EXPECT_NE(free,free2);
  
  cugraph::delete_edge_list(&G);

  //hipMemGetInfo(&free2, &total);
  //EXPECT_EQ(free,free2);
}

TEST(delete_edge_list, success2)
{
  typedef int VT;
  typedef float WT;

  std::vector<VT> src_h={0, 0, 2, 2, 2, 3, 3, 4, 4, 5}, dest_h={1, 2, 0, 1, 4, 4, 5, 3, 5, 3};
  std::vector<WT> w_h={0.50, 0.50, 0.33, 0.33, 0.33, 0.50, 0.50, 0.50, 0.50, 1.00};

  cugraph::Graph *G = new cugraph::Graph;
  gdf_column *d_src = new gdf_column, *d_dst = new gdf_column, *d_w = new gdf_column;
  //size_t free, free2, total;  
  //hipMemGetInfo(&free, &total);
  create_d_ptr(src_h, d_src);
  create_d_ptr(dest_h, d_dst);
  create_d_ptr(w_h, d_w);

  cugraph::edge_list_view(G, d_src, d_dst, d_w);
  
  //hipMemGetInfo(&free2, &total);
  //EXPECT_NE(free,free2);
  
  cugraph::delete_edge_list(G);

  //hipMemGetInfo(&free2, &total);
  //EXPECT_EQ(free,free2);

  delete G;
  delete d_src;
  delete d_dst;
  delete d_w;
}

TEST(Graph, add_transposed_adj_list)
{
  typedef int VT;
  typedef float WT;

  std::vector<VT> src_h={0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 2, 2, 2, 2, 2, 2, 2, 2, 3, 3, 3, 4, 4, 5, 5, 5, 6, 8, 8, 8, 9, 13, 14, 14, 15, 15, 18, 18, 19, 20, 20, 22, 22, 23, 23, 23, 23, 23, 24, 24, 24, 25, 26, 26, 27, 28, 28, 29, 29, 30, 30, 31, 31, 32, 1, 2, 3, 4, 5, 6, 7, 8, 10, 11, 12, 13, 17, 19, 21, 31, 2, 3, 7, 13, 17, 19, 21, 30, 3, 7, 8, 9, 13, 27, 28, 32, 7, 12, 13, 6, 10, 6, 10, 16, 16, 30, 32, 33, 33, 33, 32, 33, 32, 33, 32, 33, 33, 32, 33, 32, 33, 25, 27, 29, 32, 33, 25, 27, 31, 31, 29, 33, 33, 31, 33, 32, 33, 32, 33, 32, 33, 33};
  std::vector<VT> dest_h={1, 2, 3, 4, 5, 6, 7, 8, 10, 11, 12, 13, 17, 19, 21, 31, 2, 3, 7, 13, 17, 19, 21, 30, 3, 7, 8, 9, 13, 27, 28, 32, 7, 12, 13, 6, 10, 6, 10, 16, 16, 30, 32, 33, 33, 33, 32, 33, 32, 33, 32, 33, 33, 32, 33, 32, 33, 25, 27, 29, 32, 33, 25, 27, 31, 31, 29, 33, 33, 31, 33, 32, 33, 32, 33, 32, 33, 33, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 2, 2, 2, 2, 2, 2, 2, 2, 3, 3, 3, 4, 4, 5, 5, 5, 6, 8, 8, 8, 9, 13, 14, 14, 15, 15, 18, 18, 19, 20, 20, 22, 22, 23, 23, 23, 23, 23, 24, 24, 24, 25, 26, 26, 27, 28, 28, 29, 29, 30, 30, 31, 31, 32};
  
  cugraph::Graph *G = new cugraph::Graph;
  gdf_column *d_src = new gdf_column, *d_dst = new gdf_column;
  //size_t free, free2, free3, free4, total;  
  
  //hipMemGetInfo(&free, &total);
  
  create_d_ptr(src_h, d_src);
  create_d_ptr(dest_h, d_dst);

  //hipMemGetInfo(&free2, &total);
  //EXPECT_NE(free,free2);

  cugraph::edge_list_view(G, d_src, d_dst, nullptr);
  
  //hipMemGetInfo(&free3, &total);
  //EXPECT_EQ(free2,free3);
  //EXPECT_NE(free,free3);

  cugraph::add_transposed_adj_list(G);

  //this check doen't work on small case (false positive)
  //hipMemGetInfo(&free3, &total);
  //EXPECT_NE(free3,free2);

  std::vector<VT> off_h(G->v+1 ), ind_h(G->e);

  hipMemcpy(&off_h[0], G->transposedAdjList->offsets, sizeof(VT) * off_h.size(), hipMemcpyDeviceToHost);
  hipMemcpy(&ind_h[0], G->transposedAdjList->indices, sizeof(VT) * ind_h.size(), hipMemcpyDeviceToHost);
  size_t zero = 0;
  EXPECT_GT(off_h.size(), zero);
  EXPECT_GT(ind_h.size(), zero);
  EXPECT_EQ(off_h.size()-2, (size_t)(*(std::max_element(ind_h.begin(), ind_h.end()))));
  EXPECT_EQ(ind_h.size(), (size_t)off_h.back());

  std::sort (ind_h.begin(), ind_h.end());
  std::sort (src_h.begin(), src_h.end());

  EXPECT_EQ( eq(ind_h,src_h), 0);

  delete G;

  //hipMemGetInfo(&free4, &total);
  //EXPECT_EQ(free4,free2);
  //EXPECT_NE(free4,free);

  gdf_col_delete(d_src);
  gdf_col_delete(d_dst);

  //hipMemGetInfo(&free4, &total);
  //EXPECT_EQ(free4,free);
}

TEST(Graph, gdf_add_adjList)
{
  typedef int VT;
  typedef float WT;

  std::vector<VT> src_h={0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 2, 2, 2, 2, 2, 2, 2, 2, 3, 3, 3, 4, 4, 5, 5, 5, 6, 8, 8, 8, 9, 13, 14, 14, 15, 15, 18, 18, 19, 20, 20, 22, 22, 23, 23, 23, 23, 23, 24, 24, 24, 25, 26, 26, 27, 28, 28, 29, 29, 30, 30, 31, 31, 32, 1, 2, 3, 4, 5, 6, 7, 8, 10, 11, 12, 13, 17, 19, 21, 31, 2, 3, 7, 13, 17, 19, 21, 30, 3, 7, 8, 9, 13, 27, 28, 32, 7, 12, 13, 6, 10, 6, 10, 16, 16, 30, 32, 33, 33, 33, 32, 33, 32, 33, 32, 33, 33, 32, 33, 32, 33, 25, 27, 29, 32, 33, 25, 27, 31, 31, 29, 33, 33, 31, 33, 32, 33, 32, 33, 32, 33, 33};
  std::vector<VT> dest_h={1, 2, 3, 4, 5, 6, 7, 8, 10, 11, 12, 13, 17, 19, 21, 31, 2, 3, 7, 13, 17, 19, 21, 30, 3, 7, 8, 9, 13, 27, 28, 32, 7, 12, 13, 6, 10, 6, 10, 16, 16, 30, 32, 33, 33, 33, 32, 33, 32, 33, 32, 33, 33, 32, 33, 32, 33, 25, 27, 29, 32, 33, 25, 27, 31, 31, 29, 33, 33, 31, 33, 32, 33, 32, 33, 32, 33, 33, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 2, 2, 2, 2, 2, 2, 2, 2, 3, 3, 3, 4, 4, 5, 5, 5, 6, 8, 8, 8, 9, 13, 14, 14, 15, 15, 18, 18, 19, 20, 20, 22, 22, 23, 23, 23, 23, 23, 24, 24, 24, 25, 26, 26, 27, 28, 28, 29, 29, 30, 30, 31, 31, 32};
  std::vector<VT> off_ref_h = {0, 16, 25, 35, 41, 44, 48, 52, 56, 61, 63, 66, 67, 69, 74, 76, 78, 80, 82, 84, 87, 89, 91, 93, 98, 101, 104, 106, 110, 113, 117, 121, 127, 139, 156};

  cugraph::Graph *G = new cugraph::Graph;
  gdf_column *d_src = new gdf_column, *d_dst = new gdf_column;

  //size_t free, free2, free3, free4, total;  
  
  //hipMemGetInfo(&free, &total);
  
  create_d_ptr(src_h, d_src);
  create_d_ptr(dest_h, d_dst);

  //hipMemGetInfo(&free2, &total);
  //EXPECT_NE(free,free2);

  cugraph::edge_list_view(G, d_src, d_dst, nullptr);
  
  //hipMemGetInfo(&free3, &total);
  //EXPECT_EQ(free2,free3);
  //EXPECT_NE(free,free3);

  cugraph::add_adj_list(G);

  //this check doen't work on small case (false positive)
  //hipMemGetInfo(&free3, &total);
  //EXPECT_NE(free3,free2);

  std::vector<VT> off_h(G->v+1 ), ind_h(G->e);

  hipMemcpy(&off_h[0], G->adjList->offsets, sizeof(VT) * off_h.size(), hipMemcpyDeviceToHost);
  hipMemcpy(&ind_h[0], G->adjList->indices, sizeof(VT) * ind_h.size(), hipMemcpyDeviceToHost);
  size_t zero = 0;
  EXPECT_GT(off_h.size(), zero);
  EXPECT_GT(ind_h.size(), zero);
  EXPECT_EQ(off_h.size()-2, (size_t)(*(std::max_element(ind_h.begin(), ind_h.end()))));
  EXPECT_EQ(ind_h.size(), (size_t)off_h.back());

  std::sort (ind_h.begin(), ind_h.end());
  std::sort (dest_h.begin(), dest_h.end()); 

  EXPECT_EQ( eq(ind_h,dest_h), 0);
  EXPECT_EQ( eq(off_h,off_ref_h), 0);

  delete G;

  //hipMemGetInfo(&free4, &total);
  //EXPECT_EQ(free4,free2);
  //EXPECT_NE(free4,free);

  gdf_col_delete(d_src);
  gdf_col_delete(d_dst);

  //hipMemGetInfo(&free4, &total);
  //EXPECT_EQ(free4,free);
}
void offsets2indices(std::vector<VT> &offsets, std::vector<VT> &indices) {
  for (auto i = 0; i < offsets.size()-1; ++i) 
    for (auto j = offsets[i]; j < offsets[i+1]; ++j) 
      indices[j] = i;
}
TEST(Graph, add_edge_list)
{
  typedef int VT;
  typedef float WT;
  
  // Hard-coded Zachary Karate Club network input
  std::vector<VT> off_h = {0, 16, 25, 35, 41, 44, 48, 52, 56, 61, 63, 66, 67, 69, 74, 76, 78, 80, 82, 84, 87, 89, 91, 93, 98, 101, 104, 106, 110, 113, 117, 121, 127, 
      139, 156};
  std::vector<VT> ind_h = {1, 2, 3, 4, 5, 6, 7, 8, 10, 11, 12, 13, 17, 19, 21, 31, 0, 2, 3, 7, 13, 17, 19, 21, 30, 0, 1, 3, 7, 8, 9, 13, 27, 28, 32, 0, 1, 2, 7, 12, 13, 0, 6, 10, 0, 
      6, 10, 16, 0, 4, 5, 16, 0, 1, 2, 3, 0, 2, 30, 32, 33, 2, 33, 0, 4, 5, 0, 0, 3, 0, 1, 2, 3, 33, 32, 33, 32, 33, 5, 6, 0, 1, 32, 33, 0, 1, 33, 32, 33, 0, 1, 32, 33, 25, 27, 29, 32, 33, 
      25, 27, 31, 23, 24, 31, 29, 33, 2, 23, 24, 33, 2, 31, 33, 23, 26, 32, 33, 1, 8, 32, 33, 0, 24, 25, 28, 32, 33, 2, 8, 14, 15, 18, 20, 22, 23, 29, 30, 31, 33, 8, 9, 13, 14, 15, 
      18, 19, 20, 22, 23, 26, 27, 28, 29, 30, 31, 32};
  std::vector<WT> w_h = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
      1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
      
  cugraph::Graph *G = new cugraph::Graph;
  gdf_column *d_off = new gdf_column, *d_ind = new gdf_column, *d_w = new gdf_column;
  
  create_d_ptr(off_h, d_off);
  create_d_ptr(ind_h, d_ind);
  create_d_ptr(w_h, d_w);

  cugraph::adj_list_view(G, d_off, d_ind, d_w);

  cugraph::add_edge_list(G);

  std::vector<VT> src_h(ind_h.size()), src2_h(ind_h.size()), dest2_h(ind_h.size());
  std::vector<WT> w2_h(w_h.size());

  hipMemcpy(&src2_h[0], G->edgeList->src_indices, sizeof(VT) * ind_h.size(), hipMemcpyDeviceToHost);
  hipMemcpy(&dest2_h[0], G->edgeList->dest_indices, sizeof(VT) * ind_h.size(), hipMemcpyDeviceToHost);
  hipMemcpy(&w2_h[0], G->edgeList->edge_data, sizeof(WT) * w_h.size(), hipMemcpyDeviceToHost);
  
  offsets2indices(off_h, src_h);

  ASSERT_LE(*(std::max_element(src2_h.begin(), src2_h.end())),(VT)off_h.size()-1);
  ASSERT_GE(*(std::min_element(src2_h.begin(), src2_h.end())),off_h.front());

  ASSERT_EQ( eq(src_h,src2_h), 0);
  ASSERT_EQ( eq(ind_h,dest2_h), 0);
  ASSERT_EQ( eq(w_h,w2_h), 0);

  delete G;
  gdf_col_delete(d_off);
  gdf_col_delete(d_ind);
  gdf_col_delete(d_w);
}

TEST(Graph, get_vertex_identifiers)
{
  typedef int VT;
  typedef float WT;
  
  // Hard-coded Zachary Karate Club network input
  std::vector<VT> off_h = {0, 16, 25, 35, 41, 44, 48, 52, 56, 61, 63, 66, 67, 69, 74, 76, 78, 80, 82, 84, 87, 89, 91, 93, 98, 101, 104, 106, 110, 113, 117, 121, 127, 
      139, 156};
  std::vector<VT> ind_h = {1, 2, 3, 4, 5, 6, 7, 8, 10, 11, 12, 13, 17, 19, 21, 31, 0, 2, 3, 7, 13, 17, 19, 21, 30, 0, 1, 3, 7, 8, 9, 13, 27, 28, 32, 0, 1, 2, 7, 12, 13, 0, 6, 10, 0, 
      6, 10, 16, 0, 4, 5, 16, 0, 1, 2, 3, 0, 2, 30, 32, 33, 2, 33, 0, 4, 5, 0, 0, 3, 0, 1, 2, 3, 33, 32, 33, 32, 33, 5, 6, 0, 1, 32, 33, 0, 1, 33, 32, 33, 0, 1, 32, 33, 25, 27, 29, 32, 33, 
      25, 27, 31, 23, 24, 31, 29, 33, 2, 23, 24, 33, 2, 31, 33, 23, 26, 32, 33, 1, 8, 32, 33, 0, 24, 25, 28, 32, 33, 2, 8, 14, 15, 18, 20, 22, 23, 29, 30, 31, 33, 8, 9, 13, 14, 15, 
      18, 19, 20, 22, 23, 26, 27, 28, 29, 30, 31, 32};

  std::vector<VT> idx_h(off_h.size()-1), idx2_h(off_h.size()-1);

      
  cugraph::Graph *G = new cugraph::Graph;
  gdf_column *d_off = new gdf_column, *d_ind = new gdf_column, *col_idx = new gdf_column;
  
  create_d_ptr(off_h, d_off);
  create_d_ptr(ind_h, d_ind);
  create_d_ptr(idx2_h, col_idx);

  cugraph::adj_list_view(G, d_off, d_ind, nullptr);
  G->adjList->get_vertex_identifiers(col_idx);

  hipMemcpy(&idx2_h[0], col_idx, sizeof(VT) * idx2_h.size(), hipMemcpyDeviceToHost);
  
  std::generate(idx_h.begin(), idx_h.end(), [n = 0]() mutable {return n++;});
  
  ASSERT_EQ( eq(idx_h,idx2_h), 0);

  delete G;
  gdf_col_delete(d_off);
  gdf_col_delete(d_ind);
  gdf_col_delete(col_idx);
}

TEST(Graph, get_source_indices)
{
  typedef int VT;
  typedef float WT;
  // Hard-coded Zachary Karate Club network input
  std::vector<VT> off_h = {0, 16, 25, 35, 41, 44, 48, 52, 56, 61, 63, 66, 67, 69, 74, 76, 78, 80, 82, 84, 87, 89, 91, 93, 98, 101, 104, 106, 110, 113, 117, 121, 127, 
      139, 156};
  std::vector<VT> ind_h = {1, 2, 3, 4, 5, 6, 7, 8, 10, 11, 12, 13, 17, 19, 21, 31, 0, 2, 3, 7, 13, 17, 19, 21, 30, 0, 1, 3, 7, 8, 9, 13, 27, 28, 32, 0, 1, 2, 7, 12, 13, 0, 6, 10, 0, 
      6, 10, 16, 0, 4, 5, 16, 0, 1, 2, 3, 0, 2, 30, 32, 33, 2, 33, 0, 4, 5, 0, 0, 3, 0, 1, 2, 3, 33, 32, 33, 32, 33, 5, 6, 0, 1, 32, 33, 0, 1, 33, 32, 33, 0, 1, 32, 33, 25, 27, 29, 32, 33, 
      25, 27, 31, 23, 24, 31, 29, 33, 2, 23, 24, 33, 2, 31, 33, 23, 26, 32, 33, 1, 8, 32, 33, 0, 24, 25, 28, 32, 33, 2, 8, 14, 15, 18, 20, 22, 23, 29, 30, 31, 33, 8, 9, 13, 14, 15, 
      18, 19, 20, 22, 23, 26, 27, 28, 29, 30, 31, 32};

  std::vector<VT> src_h(ind_h.size()), src2_h(ind_h.size());
      
  cugraph::Graph *G = new cugraph::Graph;
  gdf_column *d_off = new gdf_column, *d_ind = new gdf_column, *d_src = new gdf_column;
  
  create_d_ptr(off_h, d_off);
  create_d_ptr(ind_h, d_ind);
  create_d_ptr(src2_h, d_src);

  cugraph::adj_list_view(G, d_off, d_ind, nullptr);
  G->adjList->get_source_indices(d_src);
  hipMemcpy(&src2_h[0], d_src, sizeof(VT) * G->e, hipMemcpyDeviceToHost);
  
  offsets2indices(off_h, src_h);

  ASSERT_EQ( eq(src_h,src2_h), 0);

  delete G;
  gdf_col_delete(d_off);
  gdf_col_delete(d_ind);
  gdf_col_delete(d_src);
}

TEST(Graph, gdf_column_overhead)
{
  typedef int VT;
  typedef float WT;
  
  size_t sz = 100000000;
  std::vector<VT> src_h(sz,1);
  std::vector<VT> dest_h(sz,1);

  //size_t free, free2, free3, total;  
  //hipMemGetInfo(&free, &total);

  cugraph::Graph *G = new cugraph::Graph;
  gdf_column *d_src = new gdf_column, *d_dst = new gdf_column;

  create_d_ptr(src_h, d_src);
  create_d_ptr(dest_h, d_dst);

  //hipMemGetInfo(&free2, &total);
  //EXPECT_NE(free,free2);

  // check that gdf_column_overhead < 5 per cent
  //EXPECT_LT(free-free2, 2*sz*sizeof(VT)*1.05);

  cugraph::edge_list_view(G, d_src, d_dst, nullptr);

  //hipMemGetInfo(&free3, &total);
  //EXPECT_EQ(free2,free3);
  //EXPECT_NE(free,free3);

  delete G;
  gdf_col_delete(d_src);
  gdf_col_delete(d_dst);
}

int main( int argc, char** argv )
{
    rmmInitialize(nullptr);
    testing::InitGoogleTest(&argc,argv);
    int rc = RUN_ALL_TESTS();
    rmmFinalize();
    return rc;
}
