#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithms.hpp>
#include <experimental/graph_view.hpp>
#include <iostream>
#include <patterns/count_if_v.cuh>
#include <patterns/reduce_op.cuh>
#include <patterns/update_frontier_v_push_if_out_nbr.cuh>
#include <patterns/vertex_frontier.cuh>

#include <utilities/error.hpp>
#include <vertex_partition_device.cuh>

#include <rmm/thrust_rmm_allocator.h>
#include <raft/handle.hpp>

#include <thrust/fill.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <limits>
#include <type_traits>

namespace cugraph {
namespace experimental {
namespace detail {

template <typename GraphViewType, typename PredecessorIterator>
void bfs(raft::handle_t const &handle,
         GraphViewType const &push_graph_view,
         typename GraphViewType::vertex_type *distances,
         PredecessorIterator predecessor_first,
         typename GraphViewType::vertex_type *sources,
         size_t n_sources,
         bool direction_optimizing,
         typename GraphViewType::vertex_type depth_limit,
         bool do_expensive_check)
{
  using vertex_t = typename GraphViewType::vertex_type;

  static_assert(std::is_integral<vertex_t>::value,
                "GraphViewType::vertex_type should be integral.");
  static_assert(!GraphViewType::is_adj_matrix_transposed,
                "GraphViewType should support the push model.");

  auto const num_vertices = push_graph_view.get_number_of_vertices();
  if (num_vertices == 0) { return; }
  CUGRAPH_EXPECTS(sources != nullptr, "Invalid input argument: sources cannot be null");
  CUGRAPH_EXPECTS(n_sources > 0, "Invalid input argument: input should have more than one source");

  // 1. check input arguments
  CUGRAPH_EXPECTS(
    push_graph_view.is_symmetric() || !direction_optimizing,
    "Invalid input argument: input graph should be symmetric for direction optimizing BFS.");

  // Transfer sources to the device if needed
  vertex_t *d_sources;
  rmm::device_uvector<vertex_t> d_sources_v(0, handle.get_stream());
  hipPointerAttribute_t s_att;
  CUDA_CHECK(hipPointerGetAttributes(&s_att, sources));
  if (s_att.devicePointer == nullptr) {
    d_sources_v.resize(n_sources, handle.get_stream());
    d_sources = d_sources_v.data();
    raft::copy(d_sources, sources, n_sources, handle.get_stream());

  } else {
    d_sources = sources;
  }

  if (do_expensive_check) {
    vertex_partition_device_t<GraphViewType> vertex_partition(push_graph_view);
    auto num_invalid_vertices =
      count_if_v(handle,
                 push_graph_view,
                 d_sources,
                 d_sources + n_sources,
                 [vertex_partition] __device__(auto val) {
                   return !(vertex_partition.is_valid_vertex(val) &&
                            vertex_partition.is_local_vertex_nocheck(val));
                 });
    CUGRAPH_EXPECTS(num_invalid_vertices == 0,
                    "Invalid input argument: sources have invalid vertex IDs.");
  }

  // 2. initialize distances and predecessors

  auto constexpr invalid_distance = std::numeric_limits<vertex_t>::max();
  auto constexpr invalid_vertex   = invalid_vertex_id<vertex_t>::value;

  thrust::fill(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
               distances,
               distances + push_graph_view.get_number_of_local_vertices(),
               invalid_distance);
  thrust::fill(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
               predecessor_first,
               predecessor_first + push_graph_view.get_number_of_local_vertices(),
               invalid_vertex);
  vertex_partition_device_t<GraphViewType> vertex_partition(push_graph_view);
  thrust::for_each(
    rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
    d_sources,
    d_sources + n_sources,
    [vertex_partition, distances, predecessor_first] __device__(auto v) {
      *(distances + vertex_partition.get_local_vertex_offset_from_vertex_nocheck(v)) = vertex_t{0};
    });
  // raft::print_device_vector(
  //  "distances", distances, push_graph_view.get_number_of_local_vertices(), std::cout);

  // 3. initialize BFS frontier
  enum class Bucket { cur, num_buckets };
  std::vector<size_t> bucket_sizes(static_cast<size_t>(Bucket::num_buckets),
                                   push_graph_view.get_number_of_local_vertices());
  VertexFrontier<thrust::tuple<vertex_t>,
                 vertex_t,
                 GraphViewType::is_multi_gpu,
                 static_cast<size_t>(Bucket::num_buckets)>
    vertex_frontier(handle, bucket_sizes);
  //  // if (push_graph_view.is_local_vertex_nocheck(sources)) {}
  //  vertex_frontier.get_bucket(static_cast<size_t>(Bucket::cur)).insert(d_sources, n_sources);
  if (n_sources == 1) {
    vertex_t src;
    raft::copy(&src, sources, n_sources, handle.get_stream());
    if (push_graph_view.is_local_vertex_nocheck(src)) {
      vertex_frontier.get_bucket(static_cast<size_t>(Bucket::cur)).insert(src);
    }
  }

  // 4. BFS iteration
  vertex_t depth{0};
  auto cur_local_vertex_frontier_first =
    vertex_frontier.get_bucket(static_cast<size_t>(Bucket::cur)).begin();
  auto cur_vertex_frontier_aggregate_size =
    vertex_frontier.get_bucket(static_cast<size_t>(Bucket::cur)).aggregate_size();
  while (true) {
    if (direction_optimizing) {
      CUGRAPH_FAIL("unimplemented.");
    } else {
      vertex_partition_device_t<GraphViewType> vertex_partition(push_graph_view);

      auto cur_local_vertex_frontier_last =
        vertex_frontier.get_bucket(static_cast<size_t>(Bucket::cur)).end();
      update_frontier_v_push_if_out_nbr(
        handle,
        push_graph_view,
        cur_local_vertex_frontier_first,
        cur_local_vertex_frontier_last,
        thrust::make_constant_iterator(0) /* dummy */,
        thrust::make_constant_iterator(0) /* dummy */,
        [vertex_partition, distances] __device__(
          vertex_t src, vertex_t dst, auto src_val, auto dst_val) {
          auto push = true;
          if (vertex_partition.is_local_vertex_nocheck(dst)) {
            auto distance =
              *(distances + vertex_partition.get_local_vertex_offset_from_vertex_nocheck(dst));
            if (distance != invalid_distance) { push = false; }
          }
          // FIXME: need to test this works properly if payload size is 0 (returns a tuple of size
          // 1)
          return thrust::make_tuple(push, src);
        },
        reduce_op::any<thrust::tuple<vertex_t>>(),
        distances,
        thrust::make_zip_iterator(thrust::make_tuple(distances, predecessor_first)),
        vertex_frontier,
        [depth] __device__(auto v_val, auto pushed_val) {
          auto idx = (v_val == invalid_distance)
                       ? static_cast<size_t>(Bucket::cur)
                       : VertexFrontier<thrust::tuple<vertex_t>, vertex_t>::kInvalidBucketIdx;
          return thrust::make_tuple(idx, depth + 1, thrust::get<0>(pushed_val));
        });

      auto new_vertex_frontier_aggregate_size =
        vertex_frontier.get_bucket(static_cast<size_t>(Bucket::cur)).aggregate_size() -
        cur_vertex_frontier_aggregate_size;
      if (new_vertex_frontier_aggregate_size == 0) { break; }

      cur_local_vertex_frontier_first = cur_local_vertex_frontier_last;
      cur_vertex_frontier_aggregate_size += new_vertex_frontier_aggregate_size;
    }

    depth++;
    if (depth >= depth_limit) { break; }
  }

  CUDA_TRY(hipStreamSynchronize(
    handle.get_stream()));  // this is as necessary vertex_frontier will become out-of-scope once
                            // this function returns (FIXME: should I stream sync in VertexFrontier
                            // destructor?)
}

}  // namespace detail

template <typename vertex_t, typename edge_t, typename weight_t, bool multi_gpu>
void bfs(raft::handle_t const &handle,
         graph_view_t<vertex_t, edge_t, weight_t, false, multi_gpu> const &graph_view,
         vertex_t *distances,
         vertex_t *predecessors,
         vertex_t *sources,
         size_t n_sources,
         bool direction_optimizing,
         vertex_t depth_limit,
         bool do_expensive_check)
{
  if (predecessors != nullptr) {
    detail::bfs(handle,
                graph_view,
                distances,
                predecessors,
                sources,
                n_sources,
                direction_optimizing,
                depth_limit,
                do_expensive_check);
  } else {
    detail::bfs(handle,
                graph_view,
                distances,
                thrust::make_discard_iterator(),
                sources,
                n_sources,
                direction_optimizing,
                depth_limit,
                do_expensive_check);
  }
}

// explicit instantiation

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int32_t, float, false, true> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t *sources,
                  size_t n_sources,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int32_t, double, false, true> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t *sources,
                  size_t n_sources,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int64_t, float, false, true> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t *sources,
                  size_t n_sources,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int64_t, double, false, true> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t *sources,
                  size_t n_sources,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int64_t, int64_t, float, false, true> const &graph_view,
                  int64_t *distances,
                  int64_t *predecessors,
                  int64_t *sources,
                  size_t n_sources,
                  bool direction_optimizing,
                  int64_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int64_t, int64_t, double, false, true> const &graph_view,
                  int64_t *distances,
                  int64_t *predecessors,
                  int64_t *sources,
                  size_t n_sources,
                  bool direction_optimizing,
                  int64_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int32_t, float, false, false> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t *sources,
                  size_t n_sources,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int32_t, double, false, false> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t *sources,
                  size_t n_sources,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int64_t, float, false, false> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t *sources,
                  size_t n_sources,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int64_t, double, false, false> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t *sources,
                  size_t n_sources,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int64_t, int64_t, float, false, false> const &graph_view,
                  int64_t *distances,
                  int64_t *predecessors,
                  int64_t *sources,
                  size_t n_sources,
                  bool direction_optimizing,
                  int64_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int64_t, int64_t, double, false, false> const &graph_view,
                  int64_t *distances,
                  int64_t *predecessors,
                  int64_t *sources,
                  size_t n_sources,
                  bool direction_optimizing,
                  int64_t depth_limit,
                  bool do_expensive_check);

}  // namespace experimental
}  // namespace cugraph
